// main.cu
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "utility.h"
#include "addition.h"
#include "sobel.h"
#include "memory_management.hpp"  

void addition(char *title, char *name);
void sobel(char *title, char *name);

int main(int argc, char **argv) {
    for(int i=1; i < argc; i++){
        if(strcmp(argv[i], "addition") == 0){
            addition(argv[0], argv[i]);
        }
        
        if(strcmp(argv[i], "sobel") == 0){
            sobel(argv[0], argv[i]);
        }
    }
     

    // reset device
    return (0);
}

void addition(char *title, char *name){
    printf("%s Starting %s\n", title, name);

    // set up device
    CudaDevice device(0);
    // device.printProperties();

    // set up date size of matrix
    int nx = 1<<14;
    int ny = 1<<14;
    int nxy = nx*ny;
    printf("Matrix size: nx %d ny %d\n",nx, ny);

    // allocate host memory
    HostMemory<float> h_A(nxy);
    HostMemory<float> h_B(nxy);
    HostMemory<float> hostRef(nxy);
    HostMemory<float> gpuRef(nxy);

    // initialize data at host side
    double iStart = cpuSecond();
    initialData (h_A.get(), nxy);
    initialData (h_B.get(), nxy);
    double iElaps = cpuSecond() - iStart;
    
    hostRef.memset(0);
    gpuRef.memset(0);

    // add matrix at host side for result checks
    iStart = cpuSecond();
    sumMatrixOnHost (h_A.get(), h_B.get(), hostRef.get(), nx,ny);
    iElaps = cpuSecond() - iStart;

    // allocate device global memory
    DeviceMemory<float> d_MatA(nxy);
    DeviceMemory<float> d_MatB(nxy);
    DeviceMemory<float> d_MatC(nxy);

    // transfer data from host to device
    d_MatA.copyFromHost(h_A);
    d_MatB.copyFromHost(h_B);

    // launch kernel at host side
    int dimx = 64;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);

    iStart = cpuSecond();
    sumMatrixOnGPU2D <<< grid, block >>>(d_MatA.get(), d_MatB.get(), d_MatC.get(), nx, ny);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;

    printf("sumMatrixOnGPU2D <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", grid.x,
    grid.y, block.x, block.y, iElaps);

    // copy kernel result back to host side
    d_MatC.copyToHost(gpuRef);

    // check device results
    checkResult(hostRef.get(), gpuRef.get(), nxy);
}

void sobel(char *title, char *name){
    printf("%s Starting %s\n", title, name);

    // set up device
    CudaDevice device(0);
    // device.printProperties();

    // set up image dimensions (typically smaller than matrix operations)
    int width = 1024;   // Image width
    int height = 1024;  // Image height
    int nPixels = width * height;
    printf("Image size: width %d height %d\n", width, height);

    // allocate host memory
    HostMemory<float> h_input(nPixels);
    HostMemory<float> hostRef(nPixels);
    HostMemory<float> gpuRef(nPixels);

    // initialize data at host side
    double iStart = cpuSecond();
    generateTestImage(h_input.get(), width, height); // This would typically load an actual image
    double iElaps = cpuSecond() - iStart;
    
    hostRef.memset(0);
    gpuRef.memset(0);

    // add matrix at host side for result checks
    iStart = cpuSecond();
    sobelVerticalOnHost(h_input.get(), hostRef.get(), width, height);
    iElaps = cpuSecond() - iStart;
    printf("sobelVerticalOnHost elapsed %f sec\n", iElaps);

    // allocate device global memory
    DeviceMemory<float> d_input(nPixels);
    DeviceMemory<float> d_output(nPixels);
    // transfer data from host to device
    d_input.copyFromHost(h_input);

    // launch kernel at host side
    int dimx = 32;
    int dimy = 32;
    dim3 block(dimx, dimy);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    iStart = cpuSecond();
    sobelVerticalOnGPU <<< grid, block >>>(d_input.get(), d_output.get(), width, height);
    hipDeviceSynchronize();
    iElaps = cpuSecond() - iStart;

    printf("sobelVerticalOnGPU <<<(%d,%d), (%d,%d)>>> elapsed %f sec\n", 
           grid.x, grid.y, block.x, block.y, iElaps);

    // copy kernel result back to host side
    d_output.copyToHost(gpuRef);

    // check results
    compareSobelResults(hostRef.get(), gpuRef.get(), nPixels, 1e-5f);
}