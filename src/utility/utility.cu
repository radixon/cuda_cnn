#include "hip/hip_runtime.h"
// utility.cu
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <stdbool.h>
#include "utility.h"

// Timer function - returns current time in seconds
double cpuSecond(void) {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// Initialize matrix with random data
void initialData(float *ip, int size) {
    time_t t;
    srand((unsigned int) time(&t));
    
    for (int i = 0; i < size; i++) {
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}

// Generate test image
void generateTestImage(float *image, int width, int height) {
    for(int y = 0; y < height; y++) {
        for(int x = 0; x < width; x++) {
            // Create a simple pattern with edges
            if(x < width/3) {
                image[y * width + x] = 0.0f;  // Black region
            } else if(x < 2*width/3) {
                image[y * width + x] = 128.0f;  // Gray region
            } else {
                image[y * width + x] = 255.0f;  // White region
            }
            
            // Add some vertical stripes for testing
            if((x % 20) < 5) {
                image[y * width + x] = 255.0f - image[y * width + x];
            }
        }
    }
}

// Check results between CPU and GPU
void checkResult(float *hostRef, float *gpuRef, int N) {
    double epsilon = 1.0E-8;
    bool match = true;
    int errorCount = 0;
    
    for (int i = 0; i < N; i++) {
        if (fabs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = false;
            errorCount++;
            
            // Show first 10 errors only
            if (errorCount <= 10) {
                printf("Mismatch at index %d: host=%5.2f gpu=%5.2f diff=%f\n", 
                       i, hostRef[i], gpuRef[i], fabs(hostRef[i] - gpuRef[i]));
            }
        }

        if(i < 10){
            printf("index: %d \t host element: %5.2f \t gpu element: %5.2f\n",i,hostRef[i], gpuRef[i]);
        }
    }
    
    if (match) {
        printf("Arrays match\n\n");
    } else {
        printf("Arrays do not match! Found %d errors out of %d elements.\n\n", 
               errorCount, N);
    }
}

// Function to compare results
bool compareSobelResults(float *host_result, float *gpu_result, int size, float tolerance) {
    int mismatches = 0;
    float max_diff = 0.0f;
    
    for(int i = 0; i < size; i++) {
        float diff = fabsf(host_result[i] - gpu_result[i]);
        if(diff > tolerance) {
            mismatches++;
            if(diff > max_diff) {
                max_diff = diff;
            }
        }
    }
    
    printf("Comparison Results:\n");
    printf("  Total elements: %d\n", size);
    printf("  Mismatches: %d (%.2f%%)\n", mismatches, (float)mismatches/size * 100.0f);
    printf("  Maximum difference: %f\n", max_diff);
    printf("  Tolerance: %f\n\n", tolerance);
    
    return mismatches == 0;
}

// Print matrix (useful for debugging small matrices)
void printMatrix(float *matrix, int nx, int ny, const char* name) {
    printf("\n%s Matrix (%dx%d):\n", name, nx, ny);
    
    int maxRows = (ny > 8) ? 8 : ny;
    int maxCols = (nx > 8) ? 8 : nx;
    
    for (int i = 0; i < maxRows; i++) {
        for (int j = 0; j < maxCols; j++) {
            printf("%6.2f ", matrix[i * nx + j]);
        }
        if (nx > 8) printf("...");
        printf("\n");
    }
    if (ny > 8) printf("...\n");
    printf("\n");
}

// Format bytes into human-readable format
void formatBytes(size_t bytes, char* buffer, size_t buffer_size) {
    const char* units[] = {"B", "KB", "MB", "GB", "TB"};
    int unit_index = 0;
    double size = (double)bytes;
    
    while (size >= 1024.0 && unit_index < 4) {
        size /= 1024.0;
        unit_index++;
    }
    
    if (unit_index == 0) {
        snprintf(buffer, buffer_size, "%zu %s", bytes, units[unit_index]);
    } else {
        snprintf(buffer, buffer_size, "%.2f %s", size, units[unit_index]);
    }
}

// Print basic device information (requires CUDA headers when used)
void printDeviceInfo(void) {
    printf("=== System Information ===\n");
    
    // Get current time
    time_t rawtime;
    struct tm *timeinfo;
    time(&rawtime);
    timeinfo = localtime(&rawtime);
    printf("Timestamp: %s", asctime(timeinfo));
    
    // Print some basic system info
    printf("Helper functions library loaded successfully.\n");
    printf("Timer precision: microseconds\n");
    printf("Random seed: time-based\n");
    printf("Floating point epsilon: 1.0E-8\n");
    printf("===========================\n\n");
}
